#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// CUDA kernel to add elements of two arrays
__global__
void breaker()
{
	int *ptr = (int*)0xDEADBEEF;
	*ptr = 0;
}

int main(void)
{
        hipError_t rc;
        hipStream_t s;

	rc = hipSetDevice(0);
        if (rc != hipSuccess) {
                printf("Failed to set device, %s, rc = %d\n", hipGetErrorString(rc), rc);
                return;
        }

        rc = hipStreamCreate(&s);
        if (rc != hipSuccess) {
                printf("Failed to create stream, %s, rc = %d\n", hipGetErrorString(rc), rc);
                return;
        }

        breaker<<< 1, 32, 0, s>>>();
        rc = hipStreamSynchronize(s);
        if (rc != hipSuccess) {
                printf("hipStreamSynchronize return %d, %s\n", rc, hipGetErrorString(rc));
        }
	hipDeviceSynchronize();
	cudaStreamDestory(s);
	return 0;
}
